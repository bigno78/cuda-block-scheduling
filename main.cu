#include "hip/hip_runtime.h"
#ifdef _WIN32
#include ""
#endif

#include "external/gif.h"

#include <algorithm>
#include <cmath>
#include <cstdint>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <limits>
#include <memory>
#include <string>
#include <vector>

//############################################################################//
//##                            CONSTANTS                                   ##//
//############################################################################//

namespace config
{
    constexpr int block_size = 256;
    constexpr int cell_size = 4;
    constexpr int max_nnz_per_row = 32;
    constexpr int frames = 150;
    constexpr int gif_length_seconds = 10;
}


//############################################################################//
//##                            CUDA STUFF                                  ##//
//############################################################################//

using clock64_t = long long int;

void cuda_check_result(hipError_t err, int line)
{
    if (err == hipSuccess)
        return;

    std::cerr << "[" << line << "] "
              << hipGetErrorName(err) << ": "
              << hipGetErrorString(err) << "\n"
              << "Aborting...\n";

    std::exit(1);
}

#define CUDA_CHECK_RESULT(expr) \
    cuda_check_result((expr), __LINE__)


struct DeviceDeleter
{
    void operator()(void* ptr) { hipFree(ptr); }
};

template<typename T>
using device_ptr = std::unique_ptr<T, DeviceDeleter>;


template<typename T>
device_ptr<T> device_malloc(int count)
{
    int bytes = count*sizeof(T);
    T* raw_ptr;

    CUDA_CHECK_RESULT( hipMalloc((void**)&raw_ptr, bytes) );

    return device_ptr<T>(raw_ptr);
}

template<typename T>
device_ptr<T> transfer_to_device(const std::vector<T>& vec)
{
    device_ptr<T> ptr = device_malloc<T>(vec.size());

    CUDA_CHECK_RESULT( hipMemcpy(ptr.get(), (void*)vec.data(),
                                  vec.size()*sizeof(T), hipMemcpyDefault) );

    return std::move(ptr);
}

template<typename T>
std::vector<T> transfer_to_host(const T* ptr, int count)
{
    int bytes = count*sizeof(T);
    std::vector<T> vec(count);

    CUDA_CHECK_RESULT( hipMemcpy((void*)vec.data(), ptr, bytes,
                                  hipMemcpyDefault) );

    return vec;
}


//############################################################################//
//##                            KERNEL                                      ##//
//############################################################################//

__global__ void kernel(int rows, int cols, int max_nnz_per_row,
                       const float* values, const int* column_indices,
                       const float*x, float*y, clock64_t* start_times,
                       clock64_t* end_times, int* smids)
{
    int block_id = blockIdx.y*gridDim.x + blockIdx.x;

    if (threadIdx.x == 0)
    {
        int smid;
        asm("mov.u32 %0, %%smid;" : "=r"(smid));

        start_times[block_id] = clock64();
        smids[block_id] = smid;
    }
    __syncthreads();

    int row = block_id*blockDim.x + threadIdx.x;

    if (row < rows)
    {
        float sum = 0.0f;
        for (int i = 0; i < max_nnz_per_row; ++i)
        {
            int col = column_indices[i*rows + row];
            if (0 <= col && col < cols)
                sum += values[i*rows + row]*x[col];
        }
        y[row] = sum;
    }

    __syncthreads();
    if (threadIdx.x == 0)
        end_times[block_id] = clock64();
}


//############################################################################//
//##                            DATA DEFINITIONS                            ##//
//############################################################################//

struct TimingData
{
    int grid_size_x;
    int grid_size_y;

    std::vector<clock64_t> start_times;
    std::vector<clock64_t> end_times;
    std::vector<int> smids;
};

struct ELLMatrix
{
    int rows;
    int cols;
    int max_nnz_per_row;

    std::vector<float> values;
    std::vector<int> indices;
};

struct Image
{
    int width;
    int height;

    std::vector<uint32_t> data;
};


//############################################################################//
//##                            COLOR STUFF                                 ##//
//############################################################################//

uint32_t from_rgb(uint8_t r, uint8_t g, uint8_t b)
{
    uint8_t raw_bytes[] = { r, g, b, 0 };
    uint32_t result = 0;

    std::memcpy(&result, raw_bytes, 4);

    return result;
}

namespace colors
{
    constexpr uint32_t white = 0xffffffff;

    // The color palette is taken from:
    // https://colorbrewer2.org/#type=qualitative&scheme=Set1&n=6
    std::vector<uint32_t> palette =
        [] () { return std::vector<uint32_t> {
                    from_rgb(228, 26, 28),
                    from_rgb(55, 126, 184),
                    from_rgb(77, 175, 74),
                    from_rgb(152, 78, 163),
                    from_rgb(255, 127, 0),
                    from_rgb(255, 255, 51)
                }; }();
};


//############################################################################//
//##                            GIF STUFF                                   ##//
//############################################################################//

void write_pixel(Image& image, int x, int y, uint32_t color)
{
    image.data[y*image.width + x] = color;
}

Image initialize_image(int width, int height)
{
    Image img { width, height };
    img.data.resize(width*height);

    for (int i = 0; i < width*height; ++i)
        img.data[i] = colors::white;

    return img;
}


//############################################################################//
//##                          CORE IMPLEMENTATION                           ##//
//############################################################################//

void animate_grid(TimingData& data, int frames, int seconds,
                  const std::string& filename)
{
    std::cout << "Animating the grid...\n";

    int sm_count = 1 + *max_element(data.smids.begin(), data.smids.end());
    int block_count = data.grid_size_x * data.grid_size_y;

    clock64_t max_time = std::numeric_limits<clock64_t>::max();
    std::vector<clock64_t> sm_start_times(sm_count, max_time);
    std::vector<clock64_t> sm_end_times(sm_count, 0);

    for (int i = 0; i < block_count; ++i)
    {
        int smid = data.smids[i];

        if (data.start_times[i] < sm_start_times[smid])
            sm_start_times[smid] = data.start_times[i];
        if (data.end_times[i] > sm_end_times[smid])
            sm_end_times[smid] = data.end_times[i];
    }

    clock64_t duration = 0;

    for (int i = 0; i < sm_count; ++i)
        if (sm_end_times[i] - sm_start_times[i] > duration)
            duration = sm_end_times[i] - sm_start_times[i];

    clock64_t frame_time = duration/frames;

    int dislay_width = data.grid_size_x;
    int dislay_height = data.grid_size_y;
    if (data.grid_size_y == 1)
    {
        dislay_width = std::ceil(std::sqrt(data.grid_size_x));
        dislay_height = dislay_width;
    }

    Image img = initialize_image(dislay_width*config::cell_size,
                                 dislay_height*config::cell_size);

    int centi_seconds = 100*seconds;
    int gif_delay = centi_seconds/(frames - 1);

    GifWriter g;
	GifBegin(&g, filename.c_str(), img.width, img.height, gif_delay);
    GifWriteFrame(&g, (const uint8_t*)img.data.data(), img.width, img.height,
                  gif_delay);

    for (uint64_t frame = 0; frame < frames; ++frame)
    {
        for (int i = 0; i < block_count; ++i)
        {
            int smid = data.smids[i];

            clock64_t from = sm_start_times[smid] + frame*frame_time;
            clock64_t to   = from + frame_time;

            uint32_t color = colors::white;
            if (data.start_times[i] <= to && data.end_times[i] > from)
                color = colors::palette[smid % colors::palette.size()];

            int cell_x = i % dislay_width;
            int cell_y = i / dislay_width;

            for (int x = 0; x < config::cell_size; ++x)
                for (int y = 0; y < config::cell_size; ++y)
                    write_pixel(img, cell_x*config::cell_size + x,
                                cell_y*config::cell_size + y, color);
        }

        GifWriteFrame(&g, (const uint8_t*)img.data.data(), img.width,
                      img.height, gif_delay);
    }

    GifEnd(&g);
}


ELLMatrix initialize_matrix(int rows, int cols, int max_nnz_per_row)
{
    ELLMatrix mat { rows, cols, max_nnz_per_row };

    mat.values.resize(rows*max_nnz_per_row, 1);
    mat.indices.resize(rows*max_nnz_per_row);

    int col = 0;
    for (int i = 0; i < rows; ++i)
    {
        for (int j = 0; j < max_nnz_per_row; ++j)
        {
            mat.values[j*rows + i] = (i % 3) + 1;
            mat.indices[j*rows + i] = col;
            col = (col + 1) % cols;
        }
    }

    return mat;
}

TimingData measure_block_times(int grid_size_x, int grid_size_y)
{
    int blocks = grid_size_x*grid_size_y;
    int n = blocks*config::block_size;
    int max_nnz_per_row = config::max_nnz_per_row;

    std::cout << "Creating matrix of size "
              << n << " x " << n << " with "
              << max_nnz_per_row*n << " non-zeros...\n";

    auto mat = initialize_matrix(n, n, max_nnz_per_row);
    auto x = std::vector<float>(n, 1);

    auto values_dptr = transfer_to_device(mat.values);
    auto indices_dptr = transfer_to_device(mat.indices);
    auto x_dptr = transfer_to_device(x);
    auto y_dptr = device_malloc<float>(n);

    auto start_times_dptr = device_malloc<clock64_t>(blocks);
    auto end_times_dptr = device_malloc<clock64_t>(blocks);
    auto smids_dptr = device_malloc<int>(blocks);

    std::cout << "Launching grid of size "
              << grid_size_x << " x " << grid_size_y
              << " with a total of " << blocks << " blocks...\n";

    dim3 grid_size(grid_size_x, grid_size_y);
    kernel<<<grid_size, config::block_size>>>(n, n, max_nnz_per_row,
                                      values_dptr.get(), indices_dptr.get(),
                                      x_dptr.get(), y_dptr.get(),
                                      start_times_dptr.get(),
                                      end_times_dptr.get(),
                                      smids_dptr.get());

    auto y = transfer_to_host(y_dptr.get(), n);

    for (int i = 0; i < y.size(); ++i)
    {
        float expected = max_nnz_per_row*((i % 3) + 1);
        if (y[i] != expected)
        {
            std::cout << "Invalid result at index "
                      << i << ": " << y[i] << " vs " << expected << "\n"
                      << "Aborting...\n";
            std::exit(1);
        }
    }

    std::cout << "SpMV results are correct.\n";

    TimingData res { grid_size_x, grid_size_y };
    res.start_times = transfer_to_host(start_times_dptr.get(), blocks);
    res.end_times = transfer_to_host(end_times_dptr.get(), blocks);
    res.smids = transfer_to_host(smids_dptr.get(), blocks);

    return res;
}


void print_device_info()
{
    hipDeviceProp_t prop;

    CUDA_CHECK_RESULT( hipGetDeviceProperties(&prop, 0) );

    std::cout << "Using " << prop.name << ":\n";
    std::cout << "    Number of SMs: " << prop.multiProcessorCount << "\n";
}


int main()
{
    print_device_info();
    std::cout << "\n";

    auto data_1d = measure_block_times(4096, 1);
    animate_grid(data_1d, config::frames, config::gif_length_seconds,
                 "1D_grid.gif");
    std::cout << "\n";

    auto data_2d = measure_block_times(64, 64);
    animate_grid(data_2d, config::frames, config::gif_length_seconds,
                 "2D_grid.gif");
    std::cout << "\n";

	return 0;
}
